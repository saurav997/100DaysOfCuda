#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>

__global__ void Softmax(float* input, float* output, int rows, int cols) {
    int row = threadIdx.x + blockDim.x * blockIdx.x;

    if (row < rows) {
        float max_val = -INFINITY;
        float sum = 0.0f;

        // Compute max value for numerical stability
        for (int i = 0; i < cols; i++) {
            max_val = fmaxf(max_val, input[row * cols + i]);
        }

        // Compute sum of exponentials
        for (int i = 0; i < cols; i++) {
            sum += expf(input[row * cols + i] - max_val);
        }

        // Compute final softmax values
        for (int i = 0; i < cols; i++) {
            output[row * cols + i] = expf(input[row * cols + i] - max_val) / sum;
        }
    }
}

int main() {
    int rows = 6;
    int cols = 3;
    int size = rows * cols * sizeof(float);

    float *h_input = new float[rows * cols] {1.2, 2.3, 3.4, 4.5, 5.6, 6.7, 7.8, 8.9, 9.0, 1, 2, 3, 4, 5, 6, 7, 8, 9};
    float *h_output = new float[rows * cols];

    float *d_input, *d_output;
    hipMalloc((void**)&d_input, size);
    hipMalloc((void**)&d_output, size);

    hipMemcpy(d_input, h_input, size, hipMemcpyHostToDevice);

    dim3 blockSize(256);
    dim3 gridSize((rows + blockSize.x - 1) / blockSize.x);

    Softmax<<<gridSize, blockSize>>>(d_input, d_output, rows, cols);

    hipMemcpy(h_output, d_output, size, hipMemcpyDeviceToHost);

    // Print output for verification
    std::cout << "Softmax (Global Memory) - First 5 elements: ";
    for (int i = 0; i < 5; i++) std::cout << h_output[i] << " ";
    std::cout << std::endl;

    // Free memory
    hipFree(d_input);
    hipFree(d_output);
    delete[] h_input;
    delete[] h_output;

    return 0;
}
