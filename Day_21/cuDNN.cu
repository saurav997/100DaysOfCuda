#include <iostream>
#include <hip/hip_runtime.h>
#include <hipDNN.h>

// Define input size
#define INPUT_SIZE 5

// Error handling macro for CUDA and cuDNN
#define CHECK_CUDA(call)  { 
    hipError_t err = call; 
    if (err != hipSuccess) 
    { 
        std::cerr << "CUDA error in " << __FILE__ << " line " << __LINE__ << ": " << hipGetErrorString(err) << std::endl; 
        exit(EXIT_FAILURE); 
    } 
}

#define CHECK_CUDNN(call) { 
    hipdnnStatus_t err = call; 
    if (err != HIPDNN_STATUS_SUCCESS) 
    { 
        std::cerr << "cuDNN error in " << __FILE__ << " line " << __LINE__ << ": " << hipdnnGetErrorString(err) << std::endl; 
        exit(EXIT_FAILURE); 
    } 
}

int main() {
    // 1️⃣ Initialize cuDNN
    hipdnnHandle_t cudnn;
    CHECK_CUDNN(hipdnnCreate(&cudnn));

    // 2️⃣ Define input data (ReLU will be applied to this)
    float h_input[INPUT_SIZE] = {-1.0f, 0.5f, 2.0f, -3.0f, 4.0f};  // Example values
    float h_output[INPUT_SIZE];  // To store the result

    // 3️⃣ Allocate GPU memory
    float *d_input, *d_output;
    CHECK_CUDA(hipMalloc(&d_input, INPUT_SIZE * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_output, INPUT_SIZE * sizeof(float)));

    // 4️⃣ Copy input data to GPU
    CHECK_CUDA(hipMemcpy(d_input, h_input, INPUT_SIZE * sizeof(float), hipMemcpyHostToDevice));

    // 5️⃣ Create Tensor Descriptors
    hipdnnTensorDescriptor_t inputDesc;
    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&inputDesc));
    CHECK_CUDNN(hipdnnSetTensor4dDescriptor(inputDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 1, 1, INPUT_SIZE));

    hipdnnTensorDescriptor_t outputDesc;
    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&outputDesc));
    CHECK_CUDNN(hipdnnSetTensor4dDescriptor(outputDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 1, 1, INPUT_SIZE));

    // 6️⃣ Create Activation Descriptor for ReLU
    hipdnnActivationDescriptor_t activationDesc;
    CHECK_CUDNN(hipdnnCreateActivationDescriptor(&activationDesc));
    CHECK_CUDNN(hipdnnSetActivationDescriptor(activationDesc, HIPDNN_ACTIVATION_RELU, HIPDNN_PROPAGATE_NAN, 0.0));

    // 7️⃣ Perform Activation Function (ReLU)
    float alpha = 1.0f, beta = 0.0f;
    CHECK_CUDNN(hipdnnActivationForward(cudnn, activationDesc, &alpha, inputDesc, d_input, &beta, outputDesc, d_output));

    // 8️⃣ Copy Result Back to Host
    CHECK_CUDA(hipMemcpy(h_output, d_output, INPUT_SIZE * sizeof(float), hipMemcpyDeviceToHost));

    // 9️⃣ Print Results
    std::cout << "Input:  ";
    for (int i = 0; i < INPUT_SIZE; i++) std::cout << h_input[i] << " ";
    std::cout << "\nOutput: ";
    for (int i = 0; i < INPUT_SIZE; i++) std::cout << h_output[i] << " ";
    std::cout << std::endl;

    // 🔟 Cleanup
    CHECK_CUDA(hipFree(d_input));
    CHECK_CUDA(hipFree(d_output));
    CHECK_CUDNN(hipdnnDestroyActivationDescriptor(activationDesc));
    CHECK_CUDNN(hipdnnDestroyTensorDescriptor(inputDesc));
    CHECK_CUDNN(hipdnnDestroyTensorDescriptor(outputDesc));
    CHECK_CUDNN(hipdnnDestroy(cudnn));

    return 0;
}
