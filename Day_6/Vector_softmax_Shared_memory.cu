#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>

__global__ void Softmax_Shared_Memory(const float* input, float* output, int size) {
    int numThreads = blockDim.x*gridDim.x;
    int numElementsPerThread = (size + numThreads - 1) / numThreads;
    int start_index = threadIdx.x * numElementsPerThread;
    int end_index = fmin(size, start_index + numElementsPerThread);

    if (start_index >= size) return;  // Avoid out-of-bounds threads

    // Use `extern __shared__` for dynamic shared memory allocation
    extern __shared__ float shared_data[];
    float* shared_max_val = shared_data;  // First half for max values
    float* shared_sum_exp = shared_data + numThreads;  // Second half for sum values

    float max_val = -INFINITY;
    float sum = 0.0f;

    // Step 1: Compute local max for numerical stability
    for (int i = start_index; i < end_index; i++) {
        max_val = fmaxf(max_val, input[i]);
    }
    shared_max_val[threadIdx.x] = max_val;
    __syncthreads();

    // Step 2: Find the global max using all threads
    if (threadIdx.x == 0) {
        max_val = -INFINITY;
        for (int i = 0; i < numThreads; i++) {
            max_val = fmaxf(max_val, shared_max_val[i]);
        }
        shared_max_val[0] = max_val;  // Store global max in shared memory
    }
    __syncthreads();

    // Step 3: Compute exponentials and sum
    max_val = shared_max_val[0];  // All threads use the global max
    for (int i = start_index; i < end_index; i++) {
        sum += expf(input[i] - max_val);  // Use expf for single-precision
    }
    shared_sum_exp[threadIdx.x] = sum;
    __syncthreads();

    // Step 4: Compute total sum
    if (threadIdx.x == 0) {
        sum = 0.0f;
        for (int i = 0; i < numThreads; i++) {
            sum += shared_sum_exp[i];
        }
        shared_sum_exp[0] = sum;  // Store global sum in shared memory
    }
    __syncthreads();

    // Step 5: Compute final softmax values
    sum = shared_sum_exp[0];  // All threads use the global sum
    for (int i = start_index; i < end_index; i++) {
        output[i] = expf(input[i] - max_val) / sum;  // Use expf for single-precision
    }
}

int main() {
    int size = 8;
    int space = size * sizeof(float);
    float h_input[] = {2.23, 2.33, 3.14, 4.15, 5.6, 6.17, 7.8, 1.9};

    float* h_output = new float[size];  // Use dynamic allocation

    float *d_input, *d_output;
    hipMalloc((void**)&d_input, space);
    hipMalloc((void**)&d_output, space);

    hipMemcpy(d_input, h_input, space, hipMemcpyHostToDevice);

    dim3 blockDim(4);  // Number of threads per block
    dim3 gridDim((size + blockDim.x - 1) / blockDim.x);  // Compute correct grid size

    // Specify shared memory size in kernel launch
    Softmax_Shared_Memory<<<gridDim, blockDim, 2 * blockDim.x * sizeof(float)>>>(d_input, d_output, size);

    hipMemcpy(h_output, d_output, space, hipMemcpyDeviceToHost);

    // Print results
    std::cout << "Results: " << std::endl;
    std::cout << "Original vector: ";
    for (int i = 0; i < size; i++) {
        std::cout << h_input[i] << " ";
    }
    std::cout << std::endl;
    long double sum = 0.0f;
    std::cout << "Softmax result: ";
    for (int i = 0; i < size; i++) {
        std::cout << h_output[i] << " ";
        sum += h_output[i];
    }
    std::cout << std::endl;
    std::cout<<"the sum of Softmaxes: "<<sum<<std::endl;
    //Free memory
    delete[] h_output;
    hipFree(d_input);
    hipFree(d_output);

    return 0;
}
